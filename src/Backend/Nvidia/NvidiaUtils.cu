// Copyright (c) 2019, Zpalmtree
//
// Please see the included LICENSE file for more information.

#include <iostream>
#include <map>
#include <vector>

#include "Nvidia/Argon2.h"
#include "Utilities/ColouredMsg.h"

#include <thrust/system_error.h>
#include <thrust/system/cuda/error.h>
#include <sstream>

void throw_on_cuda_error(hipError_t code, const char *file, int line)
{
    if (code == hipErrorUnknown)
    {
        std::cout << WarningMsg<std::string>("Recieved hipErrorUnknown (999) from Nvidia device. Your PC may need restarting.") << std::endl;
    }

    if (code != hipSuccess)
    {
        std::stringstream ss;
        ss << file << "(" << line << ")";
        std::string file_and_line;
        ss >> file_and_line;
        throw thrust::system_error(code, thrust::cuda_category(), file_and_line);
    }
}

int getDeviceCount()
{
    int numberDevices;
    throw_on_cuda_error(hipGetDeviceCount(&numberDevices), __FILE__, __LINE__);
    return numberDevices;
}

std::string getDeviceName(uint16_t deviceId)
{
    hipDeviceProp_t prop;
    throw_on_cuda_error(hipGetDeviceProperties(&prop, deviceId), __FILE__, __LINE__);
    return prop.name;
}

std::vector<std::tuple<std::string, bool, int>> getNvidiaDevicesActual()
{
    std::vector<std::tuple<std::string, bool, int>> devices;

    int numberDevices = getDeviceCount();

    for (int i = 0; i < numberDevices; i++)
    {
        const auto device = getDeviceName(i);

        if (device == "")
        {
            continue;
        }

        devices.push_back(std::make_tuple(device, true, i));
    }

    return devices;
}

void printNvidiaHeader()
{
    std::cout << InformationMsg<std::string>("* ") << WhiteMsg<std::string>("NVIDIA DEVICES", 25);

    int numberDevices = getDeviceCount();

    if (numberDevices == 0)
    {
        std::cout << WarningMsg<std::string>("None found") << std::endl;
        return;
    }

    std::map<std::string, std::vector<hipDeviceProp_t>> gpus;

    for (int i = 0; i < numberDevices; i++)
    {
        hipDeviceProp_t prop;
        throw_on_cuda_error(hipGetDeviceProperties(&prop, i), __FILE__, __LINE__);

        std::string deviceName = prop.name;

        gpus[deviceName].push_back(prop);
    }

    int i = 0;

    for (const auto gpu : gpus)
    {
        const auto name = gpu.first;
        const auto properties = gpu.second;

        if (i != 0)
        {
            std::cout << ", ";
        }

        size_t numDevices = properties.size();

        std::cout << SuccessMsg<size_t>(numDevices) << SuccessMsg<std::string>("x ") << SuccessMsg<std::string>(name);

        i++;
    }

    std::cout << std::endl;
}
